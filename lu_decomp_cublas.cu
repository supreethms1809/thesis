#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


void invert_device(float* src_d, float* dst_d, int n)
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int batchSize = 1;

    int *P, *INFO;

    cudacall(hipMalloc((void**)&P,n * batchSize * sizeof(int)));
    cudacall(hipMalloc((void**)&INFO,batchSize * sizeof(int)));

    int lda = n;

    float *A[] = { src_d };
    float** A_d;
    cudacall(hipMalloc((const float**)&A_d,sizeof(A)));
    cudacall(hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice));

    cublascall(hipblasSgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));

    int INFOh = 0;
    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh == n)
    {
        fprintf(stderr, "Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    float* C[] = { dst_d };
    float** C_d;
    cudacall(hipMalloc((const float**)&C_d,sizeof(C)));
    cudacall(hipMemcpy(C_d,C,sizeof(C),hipMemcpyHostToDevice));

    cublascall(hipblasSgetriBatched(handle,n,(const float**)A_d,lda,P,C_d,lda,INFO,batchSize));

    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh != 0)
    {
        fprintf(stderr, "Inversion Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    hipFree(P), hipFree(INFO), hipblasDestroy(handle);
}

void invert(float* src, float* dst, int n)
{
    float* src_d, *dst_d;

    cudacall(hipMalloc((void**)&src_d,n * n * sizeof(float)));
    cudacall(hipMemcpy(src_d,src,n * n * sizeof(float),hipMemcpyHostToDevice));
    cudacall(hipMalloc((void**)&dst_d,n * n * sizeof(float)));

    invert_device(src_d,dst_d,n);

    cudacall(hipMemcpy(dst,dst_d,n * n * sizeof(float),hipMemcpyDeviceToHost));

    hipFree(src_d);
    hipFree(dst_d);
}

void test_invert()
{
    const int n = 384;

/*    //Random matrix with full pivots
    float full_pivots[n*n] = { 0.5, 3, 4, 
                                1, 3, 10, 
                                4 , 9, 16 };
*/
    //Almost same as above matrix with first pivot zero

//    float a[n*n] = { 0, 3, 4, 
//          1, 3, 10,
//          4 , 9, 16 };

/*    float zero_pivot_col_major[n*n] = { 0, 1, 4, 
                                        3, 3, 9,
                                        4 , 10, 16 };

    float another_zero_pivot[n*n] = { 0, 3, 4, 
                                      1, 5, 6,
                                      9, 8, 2 };

    float another_full_pivot[n * n] = { 22, 3, 4, 
                                        1, 5, 6,
                                        9, 8, 2 };

    float singular[n*n] = {1,2,3,
                           4,5,6,
                           7,8,9};
*/
float *a = new float [n*n];
for(int i=0;i<n;i++)
{
for(int j=0;j<n;j++)
{
a[(i*n)+j] = ((float) rand() / 2) + 1;
}
}


    fprintf(stdout, "Input:\n\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",a[i*n+j]);
        fprintf(stdout,"\n");
    }

    fprintf(stdout,"\n\n");

    invert(a,a,n);

    fprintf(stdout, "Inverse:\n\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",a[i*n+j]);
        fprintf(stdout,"\n");
    }

}

int main()
{
    test_invert();

    int n;  scanf("%d",&n);
    return 0;
}
