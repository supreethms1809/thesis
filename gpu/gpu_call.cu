#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdlib.h>
#include <limits>
#include <ctime>
#include <string>

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define TILE_DIM 32;
#define BLOCK_ROWS 8;
#define TILE_MN 6
#define TILE_MM 4
#define TILE_WIDTH 16
#define TILE_WIDTH_INVERSE 1536

#define CHECK(call) \
{                                                                        \
        const hipError_t error = call;                                       \
        if (error != hipSuccess)                                             \
        {                                                                     \
                cout << "Error: "<<__FILE__<< " : "<<__LINE__ << endl;                      \
                cout << "code: "<<error << ", reason: " <<hipGetErrorString(error)<<endl; \
                exit(1);                                                           \
        }                                                                     \
}

__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) 
{
	__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x; 
	int ty = threadIdx.y;
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float fSum = 0;

	for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) 
	{
		if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
		{
			ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
		}
		else
		{
			ds_M[ty][tx] = 0;
		}

		if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
		{
			ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
		}
		else
		{
			ds_N[ty][tx] = 0;
		}

		__syncthreads();
       
		for (int k = 0; k < TILE_WIDTH; ++k)
		{
			fSum += ds_M[ty][k] * ds_N[k][tx];
		}
		
		__syncthreads();
	}

	if (Row < numCRows && Col < numCColumns)
	{
			C[Row*numCColumns+Col] = fSum;
	}
}


__global__ void transposeOnGPU(float *d_B, float *d_Bt, int rows, int cols)
{
	unsigned int iy = threadIdx.x + (blockIdx.x*blockDim.x);
	unsigned int ix = threadIdx.y + (blockIdx.y*blockDim.y);

	if (ix < rows && iy < cols)
	{
		d_Bt[iy*rows + ix] = d_B[ix*cols + iy];
	}
}

__global__ void check_diag_zero(float *d_m , float *d_i , const int n)
{
	int col = threadIdx.x + (blockIdx.x*blockDim.x);	
	int row = threadIdx.y + (blockIdx.y*blockDim.y);

	if(row < n && col<n)
	{
		//Checking if diagonal element is 0
                if (d_m[(n + 1)*row] == 0)
                {
                        //checking if the row is last row. If it is last row add the previous row to make it non zero
                        if (row == (n - 1))
                        {
                                d_m[(row * n) + col] = d_m[((row - 1) * n) + col] + d_m[(row * n) + col];
                                d_i[(row * n) + col] = d_i[((row - 1) * n) + col] + d_i[(row * n) + col];
                        }
			if (row != (n-1))
                        //else    //if it is not last row, add the next row.
                        {
                                d_m[(row * n) + col] = d_m[((row + 1) * n) + col] + d_m[(row * n) + col];
                                d_i[(row * n) + col] = d_i[((row + 1) * n) + col] + d_i[(row * n) + col];
                        }
                }
	}
}

__global__ void fixRow_shared(float *d_m, float *d_I,  int n, int i)
{       
        float Aii;
        int rowId = threadIdx.x;
        
        Aii = d_m[n*i+i];
        d_m[n*i+rowId] = d_m[n*i+rowId] / Aii;
        d_I[n*i+rowId] = d_I[n*i+rowId] / Aii;
}

__global__ void fixColumn_shared(float *d_m, float *d_I, const int n, const int colId)
{
	int i = threadIdx.x;
	int j = blockIdx.x;
	float AColIdj;
	__shared__ float row[TILE_WIDTH_INVERSE];
	__shared__ float rowI[TILE_WIDTH_INVERSE];

	row[i] = d_m[colId*n+i];
	rowI[i] = d_I[colId*n+i];
	AColIdj = d_m[j*n+colId];
	__syncthreads();	

	if(i < n && j < n)
	{
		if(j != colId)
		{
			d_m[j*n+i] = d_m[j*n+i] - (AColIdj*row[i]);
			d_I[j*n+i] = d_I[j*n+i] - (AColIdj*rowI[i]);
		}
	}
}

__global__ void addmu_diagonal(float *d_bbt,float *d_Zden_bbt,float mu,int row, int col)
{
	unsigned int iy = threadIdx.x + (blockIdx.x*blockDim.x);
	unsigned int ix = threadIdx.y + (blockIdx.y*blockDim.y);

	if (ix < row && iy < col)
	{
		if(ix==iy)
		{
			d_Zden_bbt[ix*col + iy] = d_bbt[ix*col + iy]+mu;
		}
		if(ix!=iy)
		{
			d_Zden_bbt[ix*col + iy] = d_bbt[ix*col + iy];
		}
	}

}

__global__ void initializeZGPU(float *d_Z, float *d_Z0, int row, int col)
{
	unsigned int iy = threadIdx.x + (blockIdx.x*blockDim.x);
	unsigned int ix = threadIdx.y + (blockIdx.y*blockDim.y);

	if (ix < row && iy < col)
	{
		d_Z0[ix*col + iy] = d_Z[ix*col + iy];
	}
}

__global__ void tempnum1Calc(float *d_tempNum1,float *d_xy,float *d_T,int row,int col)
{
	unsigned int iy = threadIdx.x + (blockIdx.x*blockDim.x);
	unsigned int ix = threadIdx.y + (blockIdx.y*blockDim.y);

	if (ix < row && iy < col)
	{
		d_tempNum1[ix*col + iy] = d_xy[ix*col + iy]-d_T[ix];
	}
}

__global__ void sumOfMatrixGPU(float *d_Znum,float *d_tempNum2,float *d_M,float *d_Y,float mu,int row,int col)
{
	unsigned int iy = threadIdx.x + (blockIdx.x*blockDim.x);
	unsigned int ix = threadIdx.y + (blockIdx.y*blockDim.y);

	if (ix < row && iy < col)
	{
		d_Znum[ix*col + iy] = d_tempNum2[ix*col + iy]+mu*d_M[ix*col + iy]+d_Y[ix*col + iy];
	}
}

__host__ void loop(float *xy,float *B,float *Bt,float *Zden,float *Z,float *Z0,float *T,float *M,float *Y,int row,int col,int row1,int col1,float mu, float *bbt)
{
	float *d_xy;
	float *d_B;
	float *d_Bt;
	float *d_bbt;
	float *d_Zden;
	float *d_Z;
	float *d_Z0;
	float *d_Zden_bbt;
	float *d_T;
	float *d_tempNum1;
	float *d_tempNum2;
	float *d_Znum;
	float *d_M;
	float *d_Y;

	const int xy_size = row*col*sizeof(float);
	const int B_size = row1*col1*sizeof(float);
	const int Bt_size = col1*row1*sizeof(float);
	const int bbt_size = row1*row1*sizeof(float);
	const int Zden_size = row1*row1*sizeof(float);
	const int Z_size = row*row1*sizeof(float);
	const int T_size = row*sizeof(float);
	const int tempNum1_size = row*col*sizeof(float);
	const int tempNum2_size = row*row1*sizeof(float);
	const int Znum_size = row*row1*sizeof(float);
	const int M_size = row*row1*sizeof(float);
	const int Y_size = row*row1*sizeof(float);


	CHECK(hipMalloc((void**)&d_xy,xy_size));
	CHECK(hipMalloc((void**)&d_B,B_size));
	CHECK(hipMalloc((void**)&d_Bt,Bt_size));
	CHECK(hipMalloc((void**)&d_bbt,bbt_size));
	CHECK(hipMalloc((void**)&d_Zden_bbt,bbt_size));
	CHECK(hipMalloc((void**)&d_Zden,Zden_size));
	CHECK(hipMalloc((void**)&d_Z,Z_size));
	CHECK(hipMalloc((void**)&d_Z0,Z_size));
	CHECK(hipMalloc((void**)&d_T,T_size));
	CHECK(hipMalloc((void**)&d_tempNum1,tempNum1_size));
	CHECK(hipMalloc((void**)&d_tempNum2,tempNum2_size));
	CHECK(hipMalloc((void**)&d_Znum,Znum_size));
	CHECK(hipMalloc((void**)&d_M,M_size));
	CHECK(hipMalloc((void**)&d_Y,Y_size));
	

	CHECK(hipMemcpy(d_xy,xy,xy_size,hipMemcpyHostToDevice));	
	CHECK(hipMemcpy(d_B,B,B_size,hipMemcpyHostToDevice));	
	CHECK(hipMemcpy(d_Bt,Bt,Bt_size,hipMemcpyHostToDevice));	
	CHECK(hipMemcpy(d_Zden,Zden,Zden_size,hipMemcpyHostToDevice));	
	CHECK(hipMemcpy(d_Z,Z,Z_size,hipMemcpyHostToDevice));	
	CHECK(hipMemcpy(d_T,T,T_size,hipMemcpyHostToDevice));	
	

	int dimx_transpose = 16;
	int dimy_transpose = 16;
	dim3 block_transpose(dimx_transpose,dimy_transpose);
	dim3 grid_transpose((col1+block_transpose.x-1)/block_transpose.x,(row1+block_transpose.y-1)/block_transpose.y);

	int dimx_bbt = 16;
	int dimy_bbt = 16;
	dim3 block_bbt(dimx_bbt,dimy_bbt);
	dim3 grid_bbt((row1+block_bbt.x-1)/block_bbt.x,(row1+block_bbt.y-1)/block_bbt.y);

	//2D grid and 2D block
        int dimx2 = row1;
        int dimy2 = 1;
        dim3 block2(dimx2,dimy2);   
        dim3 grid2(1,1); 
	
	int dimx_fixcol = row1;
        int dimy_fixcol = 1;
        dim3 block_fixcol(dimx_fixcol,dimy_fixcol);     
	dim3 grid_fixcol(row1,1); 

        int dimx3 = 32;
        int dimy3 = 32;
        dim3 block3(dimx3,dimy3); 
	dim3 grid3((row1+block3.x-1)/block3.x,(row1+block3.y-1)/block3.y);

	int dimx_ini = 16;
	int dimy_ini = 16;
	dim3 block_ini(dimx_ini,dimy_ini);
	dim3 grid_ini((row1+block_ini.x-1)/block_ini.x,(row+block_ini.y-1)/block_ini.y);

	int dimx_tempnum1 = 16;
	int dimy_tempnum1 = 16;
	dim3 block_tempnum1(dimx_tempnum1,dimy_tempnum1);
	dim3 grid_tempnum1((col+block_ini.x-1)/block_ini.x,(row+block_ini.y-1)/block_ini.y);



	cout << "2D Grid Dimension" << endl;
	cout << "\tNumber of Blocks along X dimension: " << grid_tempnum1.x << endl;
	cout << "\tNumber of Blocks along Y dimension: " << grid_tempnum1.y << endl;
	cout << "2D Block Dimension" << endl;
	cout << "\tNumber of threads along X dimension: " << block_tempnum1.x << endl;
	cout << "\tNumber of threads along Y dimension: " << block_tempnum1.y << endl;

	transposeOnGPU << <grid_transpose, block_transpose >> >(d_B, d_Bt, row1, col1);
	hipDeviceSynchronize();

	matrixMultiply<<<grid_bbt, block_bbt>>>(d_B, d_Bt, d_bbt, row1, col, col, row1, row1, row1);
	hipDeviceSynchronize();
	
	addmu_diagonal<<<grid_bbt,block_bbt>>>(d_bbt,d_Zden_bbt,mu,row1,row1);
	
	check_diag_zero << <grid3, block3 >> >(d_Zden_bbt, d_Zden, row1);
	for (int i = 0; i<row1; i++)
	{
		fixRow_shared << <grid2, block2 >> >(d_Zden_bbt, d_Zden, row1, i);
		fixColumn_shared << <grid_fixcol, block_fixcol >> >(d_Zden_bbt, d_Zden, row1, i);
	}
	hipDeviceSynchronize();
	
	for(int iter=0;iter<1;iter++)
	{
		initializeZGPU<< <grid_ini,block_ini >> >(d_Z,d_Z0,row,row1);

		//Z calculation	
		//calculateZ_preZden(Z, Zden_inv,xy, E, T, B_transpose,mu,M,Y,row,col,row1);
		tempnum1Calc<<<grid_tempnum1,block_tempnum1>>>(d_tempNum1,d_xy,d_T,row,col);
		hipDeviceSynchronize();
		matrixMultiply<<<grid_ini,block_ini>>>(d_tempNum1,d_Bt,d_tempNum2,row,col,col1,row1,row,row1);
		hipDeviceSynchronize();
		sumOfMatrixGPU<<<grid_ini,block_ini>>>(d_Znum,d_tempNum2, d_M, d_Y, mu, row, row1);
		hipDeviceSynchronize();
		matrixMultiply<<<grid_ini,block_ini>>>(d_Znum,d_Zden,d_Z,row,row1,row1,row1,row,row1);
		hipDeviceSynchronize();

		

	}


	CHECK(hipMemcpy(Bt, d_Bt, Bt_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(bbt, d_bbt, bbt_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Zden, d_Zden, Zden_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Z0, d_Z0, Z_size, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(Z, d_Z, Z_size, hipMemcpyDeviceToHost));
	//CHECK(hipMemcpy(Z, d_Znum, Z_size, hipMemcpyDeviceToHost));

	//gpu memory free
	CHECK(hipFree(d_xy));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_Bt));
	CHECK(hipFree(d_bbt));
	CHECK(hipFree(d_Zden));
	CHECK(hipFree(d_Z));
	CHECK(hipFree(d_Z0));
	CHECK(hipFree(d_Zden_bbt));
	CHECK(hipFree(d_T));
	CHECK(hipFree(d_tempNum1));
	CHECK(hipFree(d_tempNum2));
	CHECK(hipFree(d_Znum));
	CHECK(hipFree(d_M));
	CHECK(hipFree(d_Y));

}
