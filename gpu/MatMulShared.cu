#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdlib.h>
#include <limits>
#include <ctime>
#include <string>

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define CHECK(call) \
{                                                                        \
        const hipError_t error = call;                                       \
        if (error != hipSuccess)                                             \
        {                                                                     \
                cout << "Error: "<<__FILE__<< " : "<<__LINE__ << endl;                      \
                cout << "code: "<<error << ", reason: " <<hipGetErrorString(error)<<endl; \
                exit(1);                                                           \
        }                                                                     \
}

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) 
{
	__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x; 
	int ty = threadIdx.y;
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float fSum = 0;

	for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) 
	{
		if (Row < numARows && m*TILE_WIDTH+tx < numAColumns)
		{
			ds_M[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
		}
		else
		{
			ds_M[ty][tx] = 0;
		}

		if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows)
		{
			ds_N[ty][tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
		}
		else
		{
			ds_N[ty][tx] = 0;
		}

		__syncthreads();
       
		for (int k = 0; k < TILE_WIDTH; ++k)
		{
			fSum += ds_M[ty][k] * ds_N[k][tx];
		}
		
		__syncthreads();
	}

	if (Row < numCRows && Col < numCColumns)
	{
		C[Row*numCColumns+Col] = fSum;
	}
}

__host__ void gpuMultShared(float *h_A, float *h_B, float *h_C, const int A_rows, const int A_cols,const int B_rows,const int B_cols)
{
	float *d_A, *d_B, *d_C;
	int C_rows,C_cols;
	const int Matrix_A_SizeInBytes = A_rows*A_cols*sizeof(float);
	const int Matrix_B_SizeInBytes = A_cols*B_cols*sizeof(float);
	const int Matrix_C_SizeInBytes = A_rows*B_cols*sizeof(float);

	C_rows = A_rows;
	C_cols = B_cols;
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;

	CHECK(hipEventCreate(&kernel_start));
	CHECK(hipEventCreate(&kernel_stop));

	//Allocate device memory on the global memory
	CHECK(hipMalloc((void**)&d_A, Matrix_A_SizeInBytes));
	CHECK(hipMalloc((void**)&d_B, Matrix_B_SizeInBytes));
	CHECK(hipMalloc((void**)&d_C, Matrix_C_SizeInBytes));

	//transfer data from CPU Memory to GPU Memory
	CHECK(hipMemcpy(d_A, h_A, Matrix_A_SizeInBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, Matrix_B_SizeInBytes, hipMemcpyHostToDevice));

	dim3 dimGrid((C_cols-1)/TILE_WIDTH+1, (C_rows-1)/TILE_WIDTH+1, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	//Launch the GPU Kernel here
	CHECK(hipEventRecord(kernel_start));
	matrixMultiply<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, A_rows, A_cols, B_rows, B_cols, C_rows, C_cols);

	hipDeviceSynchronize();
	
	CHECK(hipEventRecord(kernel_stop));

	CHECK(hipMemcpy(h_C, d_C, Matrix_C_SizeInBytes, hipMemcpyDeviceToHost));

	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));
	CHECK(hipEventDestroy(kernel_start));
	CHECK(hipEventDestroy(kernel_stop));
}


