#include "hip/hip_runtime.h"
//rows = ny
//cols = nx
__host__ void gpuMultShared(float *h_A, float *h_B, float *h_C, const int ny, const int nx)
{
	float *d_A, *d_B, *d_C;
	const int MatrixSizeInBytes = ny*nx*sizeof(float);

	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;
	float fElapsedTime;
	float fMemoryCopyTime = 0.0f;

	CHECK(hipEventCreate(&kernel_start));
	CHECK(hipEventCreate(&kernel_stop));

	//Allocate device memory on the global memory
	CHECK(hipMalloc((void**)&d_A, MatrixSizeInBytes));
	CHECK(hipMalloc((void**)&d_B, MatrixSizeInBytes));
	CHECK(hipMalloc((void**)&d_C, MatrixSizeInBytes));

	//transfer data from CPU Memory to GPU Memory
	CHECK(hipMemcpy(d_A, h_A, MatrixSizeInBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_B, h_B, MatrixSizeInBytes, hipMemcpyHostToDevice));

	//Kernel Invoke Parameters - 2D Grid and 2D Blocks
	int dimx = 32;
	int dimy = 32;

	dim3 block(dimx, dimy);
	dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

	CHECK(hipEventRecord(kernel_start));
	SharedMult << <grid, block >> >(d_A, d_B, d_C, nx, ny);
	CHECK(hipEventRecord(kernel_stop));
	CHECK(hipEventSynchronize(kernel_stop));
	CHECK(hipEventElapsedTime(&fElapsedTime, kernel_start, kernel_stop));

	CHECK(hipMemcpy(h_C, d_C, MatrixSizeInBytes, hipMemcpyDeviceToHost));

	CHECK(hipFree(d_A));
	CHECK(hipFree(d_B));
	CHECK(hipFree(d_C));
	CHECK(hipEventDestroy(kernel_start));
	CHECK(hipEventDestroy(kernel_stop));
}

#define TILE_WIDTH 32

__global__ void SharedMult(float *g_A, float *g_B, float *g_C, const int ny, const int nx)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	//Allocate memory on the shared memory to store elements of A and B of the TILE_WIDTH x TILE_WIDTH size equal to a block
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH+2];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH+2];
	
	//Compute gloabl row and column indexes
	int col = tx + blockDim.x * bx;
	int row = ty + blockDim.y * by;

	float fSum = 0.0f;
	for (int tw_idx = 0; tw_idx < (nx / TILE_WIDTH); tw_idx++) //(nx/TILE_WIDTH)=number of phases
	{
		//Load global elements to shared memory
		s_A[ty][tx] = g_A[(row*nx) + (tw_idx*TILE_WIDTH) + tx];
		s_B[ty][tx] = g_B[(tw_idx*TILE_WIDTH + ty)*nx + col];

		__syncthreads();

		for (int k = 0; k < TILE_WIDTH; k++)
		{
			fSum += s_A[ty][k] * s_B[k][tx];
		}
		__syncthreads();
	}
	g_C[row*nx + col] = fSum;
}