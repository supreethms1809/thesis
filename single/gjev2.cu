#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdlib.h>
#include <mkl.h>
#include <mkl_lapack.h>
#include <limits>
#include <ctime>
#include <string>

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define TILE_WIDTH 32

#define CHECK(call) \
{                                                                        \
        const hipError_t error = call;                                       \
        if (error != hipSuccess)                                             \
        {                                                                     \
                cout << "Error: "<<__FILE__<< " : "<<__LINE__ << endl;                      \
                cout << "code: "<<error << ", reason: " <<hipGetErrorString(error)<<endl; \
                exit(1);                                                           \
        }                                                                     \
}


__global__ void check_diag_zero(float *d_m , float *d_i , const int n)
{
	int col = threadIdx.x + (blockIdx.x*blockDim.x);	
	int row = threadIdx.y + (blockIdx.y*blockDim.y);

	if(row < n && col<n)
	{
		//Checking if diagonal element is 0
                if (d_m[(n + 1)*row] == 0)
                {
                        //checking if the row is last row. If it is last row add the previous row to make it non zero
                        if (row == (n - 1))
                        {
                                d_m[(row * n) + col] = d_m[((row - 1) * n) + col] + d_m[(row * n) + col];
                                d_i[(row * n) + col] = d_i[((row - 1) * n) + col] + d_i[(row * n) + col];
                        }
			if (row != (n-1))
                        //else    //if it is not last row, add the next row.
                        {
                                d_m[(row * n) + col] = d_m[((row + 1) * n) + col] + d_m[(row * n) + col];
                                d_i[(row * n) + col] = d_i[((row + 1) * n) + col] + d_i[(row * n) + col];
                        }
                }
	}
}


/*
/////////////////////////////////////////method 1 - not working ///////////////////////////////////////////
__global__ void fixRow(float *d_m, float *d_I,  int n, int i)
{
	__shared__ float Ri[384];
	__shared__ float Ii[384];
	__shared__ float Aii;
	int colId = threadIdx.x;
	
	Ri[colId] = d_m[n*i+colId];
	Ii[colId] = d_I[n*i+colId];
	Aii = d_m[n*i+i];
	__syncthreads();
	
	Ri[colId] = Ri[colId] / Aii;
	Ii[colId] = Ii[colId] / Aii;
	d_m[n*i+colId] = Ri[colId];
	d_I[n*i+colId] = Ii[colId];

}

__global__ void fixColumn(float *d_m, float *d_I, const int n, const int colId)
{
	int i = threadIdx.x;
	int j = blockIdx.x;
	
	__shared__ float col[384];
	__shared__ float Icol[384];

	__shared__ float AColIdj;

	__shared__ float colj[384];
	__shared__ float Icolj[384];

	if(i < n && j < n)
	{
	col[i] = d_m[i*n+colId];
	Icol[i] = d_I[i*n+colId];
//	printf("threadId = %d\n",i);
//	printf("blockId = %d\n",j);
	//if(col[i] != 0)
	//{
		colj[i] = d_m[i*n+j];
		Icolj[i] = d_I[i*n+j];
		AColIdj = d_m[colId * n +j];
		//AColIdj = d_m[colId + n *j];
		__syncthreads();
		if(j != colId)
		{
		Icolj[i] = Icolj[i] - AColIdj * Icol[i];
		if(i != colId)
		{
			colj[i] = colj[i] - AColIdj * col[i];

		}
		}
		d_m[i*n+j] = colj[i];
		d_I[i*n+j] = Icolj[i];
	
	//}
	}
}
*/


///////////////////////////////////method 2 - working /////////////////////////////////
__global__ void nodiag_normalize(float *A, float *I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float temp = A[i*n + i];
	__shared__ float col[TILE_WIDTH][TILE_WIDTH+2];
	__shared__ float Icol[TILE_WIDTH][TILE_WIDTH+2];

	if (x < n && y < n)
	{
		Icol[ty][tx] = I[x*n + y];
		col[ty][tx] = A[x*n + y];
		__syncthreads();
		if (x == i && x != y)
		{
			//I[x*n + y] /= A[i*n + i];
			//A[x*n + y] /= A[i*n + i];
//			I[x*n + y] /= temp;
//			A[x*n + y] /= temp;
			Icol[ty][tx] /= temp;
			col[ty][tx] /= temp;
		}
		I[x*n + y] = Icol[ty][tx];
		A[x*n + y] = col[ty][tx];
		__syncthreads();
	}
	
}

__global__ void diag_normalize(float *A, float *I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	__shared__ float col[TILE_WIDTH][TILE_WIDTH+2];
	__shared__ float Icol[TILE_WIDTH][TILE_WIDTH+2];

	float temp = A[i*n + i];
	if (x < n && y < n)
	{
		Icol[ty][tx] = I[x*n + y];
		col[ty][tx] = A[x*n + y];
		
		__syncthreads();
		if (x == y && x == i)
		{
			//I[x*n + y] /= A[i*n + i];
			//A[x*n + y] /= A[i*n + i];
//			I[x*n + y] /= temp;
//			A[x*n + y] /= temp;
			Icol[ty][tx] /= temp;
			col[ty][tx] /= temp;

		}
		I[x*n + y] = Icol[ty][tx];
		A[x*n + y] = col[ty][tx];

	}
}


__global__ void gaussjordan_old(float *A, float *I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	__shared__ float col[TILE_WIDTH][TILE_WIDTH+2];
	__shared__ float Icol[TILE_WIDTH][TILE_WIDTH+2];
	float A_temp = A[x*n + i];
	
	if (x < n && y < n)
	{
		Icol[ty][tx] = I[x*n + y];
		col[ty][tx] = A[x*n + y];
		
		__syncthreads();
		if (x != i)
		{
//			I[x*n + y] -= I[i*n + y] * A[x*n + i];
			Icol[ty][tx] -= I[i*n + y] * A_temp;
			if (y != i)
			{
//				A[x*n + y] -= A[i*n + y] * A[x*n + i];
				col[ty][tx] -= A[i*n + y] * A_temp;
			}	 
		}
		I[x*n + y] = Icol[ty][tx];
		A[x*n + y] = col[ty][tx];
	}

}


/*
//////////////////////////////method 3 - not working///////////////////////////
 __global__ void gaussjordan(float *A,  float *I,int n, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    float P;

    if(x<n && y<n)
        if(x>i)
	{ // this limits operation to rows below the pivot point
            P=A[x*n+i]/A[i*n+i];
            I[x*n+y] -= I[i*n+y]*P;  // apply for every row member
            if(y>=i)
		{ //limits  to row members to the right of the pivot
                A[x*n+y] -= A[i*n+y]*P;  // apply only to members right of pivot
		}
        }
 }

 __global__ void dev(float *d_A,  float *dI, int h)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x<h && y<h)
        if(d_A[x*h+x]!=0){
            dI[x*h+y]  /= d_A[x*h+x];
            d_A[x*h+y] /= d_A[x*h+x];
        }
    __syncthreads();

}
*/

__host__ void gpuInverseOfMatrix(float *h_matrix,float *h_iden_mat, int col)
{
	float *d_matrix,*d_iden_mat;
	const int MatSizeInBytes = col*col*sizeof(float);

	hipError_t hipSetDevice(int device);
        hipSetDevice(0);
               
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;

	CHECK(hipEventCreate(&kernel_start));
	CHECK(hipEventCreate(&kernel_stop));

	//Allocate device memory on the global memory
	CHECK(hipMalloc((void**)&d_matrix, MatSizeInBytes));
	CHECK(hipMalloc((void**)&d_iden_mat, MatSizeInBytes));

	CHECK(hipMemcpy(d_matrix, h_matrix, MatSizeInBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_iden_mat, h_iden_mat, MatSizeInBytes, hipMemcpyHostToDevice));

	//2D grid and 2D block
	int dimx1 = 32;
	int dimy1 = 32;
	dim3 block1(dimx1,dimy1);
	dim3 grid1((col+block1.x-1)/block1.x,(col+block1.y-1)/block1.y);

        int dimx2 = col;
        int dimy2 = 1;
        dim3 block2(dimx2,dimy2);                                                           
        dim3 grid2(1,1); 

        int dimx3 = 32;
        int dimy3 = 32;
        dim3 block3(dimx3,dimy3);                                                           
	dim3 grid3((col+block3.x-1)/block3.x,(col+block3.y-1)/block3.y);

//	cout << "\t2D Grid Dimension" << endl;
//	cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
//	cout << "\tNumber of Blocks along Y dimension: " << grid.y << endl;
//	cout << "\t2D Block Dimension" << endl;
//	cout << "\tNumber of threads along X dimension: " << block.x << endl;
//	cout << "\tNumber of threads along Y dimension: " << block.y << endl;

	CHECK(hipEventRecord(kernel_start));
	check_diag_zero << <grid3, block3 >> >(d_matrix, d_iden_mat, col);
	//for (int i = 0; i<col; i++)
	//{
	//	fixRow << <grid2, block2 >> >(d_matrix, d_iden_mat, col, i);
	//	//fixColumn << <grid3, block3 >> >(d_matrix, d_iden_mat, col, i);
	//	gaussjordan << <grid3, block3 >> >(d_matrix, d_iden_mat, col, i);
	//}
	for (int i = 0; i<col; i++)
	{
		nodiag_normalize << <grid3, block3 >> >(d_matrix, d_iden_mat, col, i);
		diag_normalize << <grid3, block3 >> >(d_matrix, d_iden_mat, col, i);
		//CHECK(hipDeviceSynchronize());
		gaussjordan_old << <grid3, block3 >> >(d_matrix, d_iden_mat, col, i);
		//set_zero << <grid, block >> >(d_matrix, d_iden_mat, col, i);
	}


		//dev << <grid3, block3 >> >(d_matrix, d_iden_mat, col);
	CHECK(hipDeviceSynchronize());
	CHECK(hipEventRecord(kernel_stop));
	CHECK(hipEventSynchronize(kernel_stop));

	CHECK(hipMemcpy(h_matrix, d_matrix, MatSizeInBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_iden_mat, d_iden_mat, MatSizeInBytes, hipMemcpyDeviceToHost));

	CHECK(hipFree(d_matrix));
	CHECK(hipFree(d_iden_mat));
	CHECK(hipEventDestroy(kernel_start));
	CHECK(hipEventDestroy(kernel_stop));
	CHECK(hipDeviceReset());

}



