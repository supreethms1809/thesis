#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdlib.h>
#include <mkl.h>
#include <mkl_lapack.h>
#include <limits>
#include <ctime>
#include <string>
#include <chrono>

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define TILE_WIDTH 384

#define CHECK(call) \
{                                                                        \
        const hipError_t error = call;                                       \
        if (error != hipSuccess)                                             \
        {                                                                     \
                cout << "Error: "<<__FILE__<< " : "<<__LINE__ << endl;                      \
                cout << "code: "<<error << ", reason: " <<hipGetErrorString(error)<<endl; \
                exit(1);                                                           \
        }                                                                     \
}


__host__ void gpuProx_2norm(float *Q, float *M, float *C, float constant, int row, int col, int data_size)
{
	float *d_Q,*d_M,*d_C;
	const int MatSizeInBytes = row*col*sizeof(float);
	const int CsizeInBytes = col*sizeof(float);

	//memory allocation on GPU
	CHECK(hipMalloc((void**)&d_Q,MatSizeInBytes));
	CHECK(hipMalloc((void**)&d_M,u_size));
	CHECK(hipMalloc((void**)&d_C,sig_size));
	
	//data copy into GPU memory
	CHECK(hipMemcpy(d_Q,Q,MatSizeInBytes,hipMemcpyHostToDevice));

	//2D grid and 2D block
	int dimx = 1;
	int dimy = 1;
	dim3 block(dimx,dimy);
	dim3 grid((col+block.x-1)/block.x,1);
	cout << "threads in a block "<<block<<endl;
	cout << "blocks in a grid "<<grid <<endl;

	//copy back data from GPU
	CHECK(hipMemcpy(M,d_M,MatSizeInBytes,hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(C,d_C,MatSizeInBytes,hipMemcpyDeviceToHost));
	CHECK(hipFree(d_Q));
	CHECK(hipFree(d_M));
	CHECK(hipFree(d_C));


}
