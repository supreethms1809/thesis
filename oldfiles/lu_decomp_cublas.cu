#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


void invert_device(double* src_d, double* dst_d, int n)
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int batchSize = 1;

    int *P, *INFO;

    cudacall(hipMalloc((void**)&P,n * batchSize * sizeof(int)));
    cudacall(hipMalloc((void**)&INFO,batchSize * sizeof(int)));

    int lda = n;

    double *A[] = { src_d };
    double** A_d;
    cudacall(hipMalloc((const double**)&A_d,sizeof(A)));
    cudacall(hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice));

    cublascall(hipblasDgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));

    int INFOh = 0;
    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh == n)
    {
        fprintf(stderr, "Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    double* C[] = { dst_d };
    double** C_d;
    cudacall(hipMalloc((const double**)&C_d,sizeof(C)));
    cudacall(hipMemcpy(C_d,C,sizeof(C),hipMemcpyHostToDevice));

    cublascall(hipblasDgetriBatched(handle,n,(const double**)A_d,lda,P,C_d,lda,INFO,batchSize));

    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh != 0)
    {
        fprintf(stderr, "Inversion Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    hipFree(P), hipFree(INFO), hipblasDestroy(handle);
}

void invert(double* src, double* dst, int n)
{
    double* src_d, *dst_d;

    cudacall(hipMalloc((void**)&src_d,n * n * sizeof(double)));
    cudacall(hipMemcpy(src_d,src,n * n * sizeof(double),hipMemcpyHostToDevice));
    cudacall(hipMalloc((void**)&dst_d,n * n * sizeof(double)));

    invert_device(src_d,dst_d,n);

    cudacall(hipMemcpy(dst,dst_d,n * n * sizeof(double),hipMemcpyDeviceToHost));

    hipFree(src_d);
    hipFree(dst_d);
}

void test_invert()
{
    const int n = 3;

/*    //Random matrix with full pivots
    double full_pivots[n*n] = { 0.5, 3, 4, 
                                1, 3, 10, 
                                4 , 9, 16 };
*/
    //Almost same as above matrix with first pivot zero

    double a[n*n] = { 0, 3, 4, 
          1, 3, 10,
          4 , 9, 16 };

/*    double zero_pivot_col_major[n*n] = { 0, 1, 4, 
                                        3, 3, 9,
                                        4 , 10, 16 };

    double another_zero_pivot[n*n] = { 0, 3, 4, 
                                      1, 5, 6,
                                      9, 8, 2 };

    double another_full_pivot[n * n] = { 22, 3, 4, 
                                        1, 5, 6,
                                        9, 8, 2 };

    double singular[n*n] = {1,2,3,
                           4,5,6,
                           7,8,9};
*/
/*double *a = new double [n*n];
for(int i=0;i<n;i++)
{
for(int j=0;j<n;j++)
{
a[(i*n)+j] = ((double) rand() / 2) + 1;
}
}
*/

    fprintf(stdout, "Input:\n\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",a[i*n+j]);
        fprintf(stdout,"\n");
    }

    fprintf(stdout,"\n\n");

    invert(a,a,n);

    fprintf(stdout, "Inverse:\n\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",a[i*n+j]);
        fprintf(stdout,"\n");
    }

}

int main()
{
    test_invert();

    return 0;
}
