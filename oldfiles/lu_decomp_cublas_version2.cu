#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


void invert_device(float* src_d, float* dst_d, int n)
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int batchSize = 1;

    int *P, *INFO;

    cudacall(hipMalloc<int>(&P,17 * batchSize * sizeof(int)));
    cudacall(hipMalloc<int>(&INFO,batchSize * sizeof(int)));

    int lda = 17;

    float *A[] = { src_d };
    float** A_d;
    cudacall(hipMalloc<float*>(&A_d,sizeof(A)));
    cudacall(hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice));

    cublascall(hipblasSgetrfBatched(handle,17,A_d,lda,P,INFO,batchSize));

    int INFOh = 0;
    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh == 17)
    {
        fprintf(stderr, "Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    float* C[] = { dst_d };
    float** C_d;
    cudacall(hipMalloc<float*>(&C_d,sizeof(C)));
    cudacall(hipMemcpy(C_d,C,sizeof(C),hipMemcpyHostToDevice));

    cublascall(hipblasSgetriBatched(handle,n,A_d,lda,P,C_d,n,INFO,batchSize));

    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh != 0)
    {
        fprintf(stderr, "Inversion Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    hipFree(P), hipFree(INFO), hipblasDestroy(handle);
}

void invert(float* src, float* dst, int n)
{
    float* src_d, *dst_d;

    cudacall(hipMalloc<float>(&src_d,17 * 17 * sizeof(float)));
    cudacall(hipMemcpy(src_d,src,17 * 17 * sizeof(float),hipMemcpyHostToDevice));
    cudacall(hipMalloc<float>(&dst_d,n * n * sizeof(float)));

    invert_device(src_d,dst_d,n);

    cudacall(hipMemcpy(dst,dst_d,n * n * sizeof(float),hipMemcpyDeviceToHost));

    hipFree(src_d), hipFree(dst_d);
}

void test_invert()
{
    const int n = 3;

    //Random matrix with full pivots
/*    float full_pivots[n*n] = { 0.5, 3, 4,
                                1, 3, 10,
                                4 , 9, 16 };

    //Almost same as above matrix with first pivot zero
    float zero_pivot[n*n] = { 0, 3, 4,
                              1, 3, 10,
                              4 , 9, 16 };

    float zero_pivot_col_major[n*n] = { 0, 1, 4,
                                        3, 3, 9,
                                        4 , 10, 16 };
*/
    float zero_pivot_war[17*17] = {
                                        0,3,4,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                                        1,3,10,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                                        4,9,16,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
                                        0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,
                                        0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,0,
                                        0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,0,
                                        0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,0,
                                        0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0,
                                        0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,
                                        0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,0,
                                        0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,0,
                                        0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,0,
                                        0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,
                                        0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,
                                        0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,
                                        0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,
                                        0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1 };
/*
    float another_zero_pivot[n*n] = { 0, 3, 4,
                                      1, 5, 6,
                                      9, 8, 2 };

    float another_full_pivot[n * n] = { 22, 3, 4,
                                        1, 5, 6,
                                        9, 8, 2 };

    float singular[n*n] = {1,2,3,
                           4,5,6,
                           7,8,9};
*/
    float result[n*n];

    //Select matrix by setting "a"
    float* a = zero_pivot_war;

    fprintf(stdout, "Input:\n\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",a[i*17+j]);
        fprintf(stdout,"\n");
    }

    fprintf(stdout,"\n\n");

    invert(a,result,n);

    fprintf(stdout, "Inverse:\n\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
            fprintf(stdout,"%f\t",result[i*n+j]);
        fprintf(stdout,"\n");
    }

}

int main()
{
    test_invert();

//    int n;  scanf("%d",&n);
    return 0;
}
