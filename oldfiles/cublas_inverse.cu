#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


void invert_device(double* src_d, double* dst_d, int n)
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int batchSize = 1;

    int *P, *INFO;

    cudacall(hipMalloc((void**)&P,n * batchSize * sizeof(int)));
    cudacall(hipMalloc((void**)&INFO,batchSize * sizeof(int)));

    int lda = n;

    double *A[] = { src_d };
    double** A_d;
    cudacall(hipMalloc((const double**)&A_d,sizeof(A)));
    cudacall(hipMemcpy(A_d,A,sizeof(A),hipMemcpyHostToDevice));

    cublascall(hipblasDgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize));

    int INFOh = 0;
    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh == n)
    {
        fprintf(stderr, "Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    double* C[] = { dst_d };
    double** C_d;
    cudacall(hipMalloc((const double**)&C_d,sizeof(C)));
    cudacall(hipMemcpy(C_d,C,sizeof(C),hipMemcpyHostToDevice));

    cublascall(hipblasDgetriBatched(handle,n,(const double**)A_d,lda,P,C_d,lda,INFO,batchSize));

    cudacall(hipMemcpy(&INFOh,INFO,sizeof(int),hipMemcpyDeviceToHost));

    if(INFOh != 0)
    {
        fprintf(stderr, "Inversion Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }

    hipFree(P), hipFree(INFO), hipblasDestroy(handle);
}

void invert(double* src, double* dst, int n)
{
    double* src_d, *dst_d;

    cudacall(hipMalloc((void**)&src_d,n * n * sizeof(double)));
    cudacall(hipMemcpy(src_d,src,n * n * sizeof(double),hipMemcpyHostToDevice));
    cudacall(hipMalloc((void**)&dst_d,n * n * sizeof(double)));

    invert_device(src_d,dst_d,n);

    cudacall(hipMemcpy(dst,dst_d,n * n * sizeof(double),hipMemcpyDeviceToHost));

    hipFree(src_d);
    hipFree(dst_d);
}

