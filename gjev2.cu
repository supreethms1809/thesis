#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <math.h>
#include <stdlib.h>
#include <mkl.h>
#include <mkl_lapack.h>
#include <limits>
#include <ctime>
#include <string>

#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define CHECK(call) \
{                                                                        \
        const hipError_t error = call;                                       \
        if (error != hipSuccess)                                             \
        {                                                                     \
                cout << "Error: "<<__FILE__<< " : "<<__LINE__ << endl;                      \
                cout << "code: "<<error << ", reason: " <<hipGetErrorString(error)<<endl; \
                exit(1);                                                           \
        }                                                                     \
}


__global__ void check_diag_zero(double *d_m , double *d_i , const int n)
{
	int col = threadIdx.x + (blockIdx.x*blockDim.x);	
	int row = threadIdx.y + (blockIdx.y*blockDim.y);

	if(row < n && col<n)
	{
		//Checking if diagonal element is 0
                if (d_m[(n + 1)*row] == 0)
                {
                        //checking if the row is last row. If it is last row add the previous row to make it non zero
                        if (row == (n - 1))
                        {
                                d_m[(row * n) + col] = d_m[((row - 1) * n) + col] + d_m[(row * n) + col];
                                d_i[(row * n) + col] = d_i[((row - 1) * n) + col] + d_i[(row * n) + col];
                        }
			if (row != (n-1))
                        //else    //if it is not last row, add the next row.
                        {
                                d_m[(row * n) + col] = d_m[((row + 1) * n) + col] + d_m[(row * n) + col];
                                d_i[(row * n) + col] = d_i[((row + 1) * n) + col] + d_i[(row * n) + col];
                        }
                }
	}
}

__global__ void fixRow(double *d_m, double *d_I,  int n, int i)
{
	__shared__ double Ri[384];
	__shared__ double Ii[384];
	__shared__ double Aii;
	int colId = threadIdx.x;
	
	Ri[colId] = d_m[n*i+colId];
	Ii[colId] = d_I[n*i+colId];
	Aii = d_m[n*i+i];
	__syncthreads();
	
	Ri[colId] = Ri[colId] / Aii;
	Ii[colId] = Ii[colId] / Aii;
	d_m[n*i+colId] = Ri[colId];
	d_I[n*i+colId] = Ii[colId];

}

__global__ void fixColumn(double *d_m, double *d_I, const int n, const int colId)
{
	int i = threadIdx.x;
	int j = blockIdx.x;
	
	__shared__ double col[384];
	__shared__ double Icol[384];

	__shared__ double AColIdj;

	__shared__ double colj[384];
	__shared__ double Icolj[384];

	if(i < n && j < n)
	{
	col[i] = d_m[i*n+colId];
	Icol[i] = d_I[i*n+colId];
//	printf("threadId = %d\n",i);
//	printf("blockId = %d\n",j);
	//if(col[i] != 0)
	//{
		colj[i] = d_m[i*n+j];
		Icolj[i] = d_I[i*n+j];
		AColIdj = d_m[colId * n +j];
		//AColIdj = d_m[colId + n *j];
		__syncthreads();
		if(j != colId)
		{
		Icolj[i] = Icolj[i] - AColIdj * Icol[i];
		if(i != colId)
		{
			colj[i] = colj[i] - AColIdj * col[i];

		}
		}
		d_m[i*n+j] = colj[i];
		d_I[i*n+j] = Icolj[i];
	
	//}
	}
}


__host__ void gpuInverseOfMatrix(double *h_matrix,double *h_iden_mat, int col)
{
	double *d_matrix,*d_iden_mat;
	const int MatSizeInBytes = col*col*sizeof(double);

	hipError_t hipSetDevice(int device);
        hipSetDevice(0);
               
	hipEvent_t kernel_start;
	hipEvent_t kernel_stop;

	CHECK(hipEventCreate(&kernel_start));
	CHECK(hipEventCreate(&kernel_stop));

	//Allocate device memory on the global memory
	CHECK(hipMalloc((void**)&d_matrix, MatSizeInBytes));
	CHECK(hipMalloc((void**)&d_iden_mat, MatSizeInBytes));

	CHECK(hipMemcpy(d_matrix, h_matrix, MatSizeInBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_iden_mat, h_iden_mat, MatSizeInBytes, hipMemcpyHostToDevice));

	//2D grid and 2D block
	int dimx1 = 32;
	int dimy1 = 32;
	dim3 block1(dimx1,dimy1);
	dim3 grid1((col+block1.x-1)/block1.x,(col+block1.y-1)/block1.y);

        int dimx2 = col;
        int dimy2 = 1;
        dim3 block2(dimx2,dimy2);                                                           
        dim3 grid2(1,1); 

        int dimx3 = col;
        int dimy3 = 1;
        dim3 block3(dimx3,dimy3);                                                           
        dim3 grid3(col,1); 

//	cout << "\t2D Grid Dimension" << endl;
//	cout << "\tNumber of Blocks along X dimension: " << grid.x << endl;
//	cout << "\tNumber of Blocks along Y dimension: " << grid.y << endl;
//	cout << "\t2D Block Dimension" << endl;
//	cout << "\tNumber of threads along X dimension: " << block.x << endl;
//	cout << "\tNumber of threads along Y dimension: " << block.y << endl;

	CHECK(hipEventRecord(kernel_start));
	check_diag_zero << <grid1, block1 >> >(d_matrix, d_iden_mat, col);
	for (int i = 0; i<col; i++)
	{
		fixRow << <grid2, block2 >> >(d_matrix, d_iden_mat, col, i);
		fixColumn << <grid3, block3 >> >(d_matrix, d_iden_mat, col, i);
	}

//	check_diag_zero << <grid, block >> >(d_matrix, d_iden_mat,col);
//	CHECK(hipDeviceSynchronize());
//	for(int i = 0; i < col; i++)
//	{
//		NaiveInverse << <grid, block >> >(d_matrix, d_iden_mat,col,i);		
//	}	
//	CHECK(hipDeviceSynchronize());
	CHECK(hipEventRecord(kernel_stop));
	CHECK(hipEventSynchronize(kernel_stop));

	CHECK(hipMemcpy(h_matrix, d_matrix, MatSizeInBytes, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_iden_mat, d_iden_mat, MatSizeInBytes, hipMemcpyDeviceToHost));

	CHECK(hipFree(d_matrix));
	CHECK(hipFree(d_iden_mat));
	CHECK(hipEventDestroy(kernel_start));
	CHECK(hipEventDestroy(kernel_stop));
	CHECK(hipDeviceReset());

}



